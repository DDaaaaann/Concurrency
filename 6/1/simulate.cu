#include "hip/hip_runtime.h"
/*
 * simulate.cu
 *
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <iostream>

#include "file.h"
#include "timer.h"
#include "simulate.h"
#include "simulate.h"

#define BLOCK_SIZE 512

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error: %s\n", hipGetErrorString(result));
        exit(1);
    }
}

__global__ void calculate_next(double *dev_old, double *dev_cur,
        double *dev_new, int t_max, int timestep) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int t_id = threadIdx.x;
    if (t_id < 2 || t_id > BLOCK_SIZE - 2) {
        printf("IIIII caaaaaaaaannnnnnnnnnnn beeeeeeeeee %d\n", t_id);
    }


    if (i >= t_max) {
        printf("too large i = %d   t_id = %d timestep = %d\n", blockIdx.x, t_id, timestep);
        return;
    }

    printf("Got past sizechecking i = %d   t_id = %d tp= %d\n", blockIdx.x, t_id, timestep);

    __shared__ double s_cur[BLOCK_SIZE];

    printf("Got past creating shared thing i = %d   t_id = %d  tp=%d\n", blockIdx.x, t_id, timestep);

    s_cur[t_id] = dev_cur[i];
    printf("Got past filling shared i = %d   t_id = %d  tp=%d\n", blockIdx.x, t_id, timestep);

    __syncthreads();

    printf("Got past threadsyncing i = %d   t_id = %d  tp=%d\n", blockIdx.x, t_id, timestep);

    if (t_id == 0) {
        printf("Got in first if i = %d   t_id = %d  tp=%d\n", blockIdx.x, t_id, timestep);
        dev_new[i] = 2 * s_cur[t_id] - dev_old[i] + 0.2 * (dev_cur[i - 1] -
                (2 * s_cur[t_id] - s_cur[t_id + 1]));
    }
    else if (t_id == BLOCK_SIZE - 1) {
        printf("Got in second if i = %d   t_id = %d  tp=%d\n", blockIdx.x, t_id, timestep);
        dev_new[i] = 2 * s_cur[t_id] - dev_old[i] + 0.2 * (dev_cur[i - 1] -
                (2 * s_cur[t_id] - s_cur[t_id + 1]));
    }
    else {
        printf("Got in third if i = %d   t_id = %d  tp=%d\n", blockIdx.x, t_id, timestep);
        dev_new[i] = 2 * s_cur[t_id] - dev_old[i] + 0.2 * (dev_cur[i - 1] -
                (2 * s_cur[t_id] - s_cur[t_id + 1]));
    }

    printf("Got past calculating i = %d   t_id = %d  tp=%d\n", blockIdx.x, t_id, timestep);

}

/*
 * Executes the entire simulation.
 *
 * Implement your code here.
 *
 * i_max: how many data points are on a single wave
 * t_max: how many iterations the simulation should run
 * block_size: how many threads to use (excluding the main threads)
 * old_array: array of size i_max filled with data for t-1
 * current_array: array of size i_max filled with data for t
 * next_array: array of size i_max. You should fill this with t+1
 */
double *simulate(const int i_max, const int t_max, const int block_size,
        double *old_array, double *current_array, double *next_array)
{
    printf("Got in simulation \n");
    double *dev_old, *dev_cur, *dev_new;
    printf("Got past declaring variables \n");

    // allocate the vectors on the GPU
    checkCudaCall(hipMalloc(&dev_old, t_max * sizeof(double)));
    checkCudaCall(hipMalloc(&dev_cur, t_max * sizeof(double)));
    checkCudaCall(hipMalloc(&dev_new, t_max * sizeof(double)));

    printf("Got past hipMalloc \n");

    // add events to maxe the time correct
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("Got past eventCreate \n");

    // copy data to the vectors
    checkCudaCall(hipMemcpy(dev_old, old_array, t_max * sizeof(double),
            hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(dev_cur, current_array, t_max * sizeof(double),
            hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(dev_new, next_array, t_max * sizeof(double),
            hipMemcpyHostToDevice));

    printf("Got past filling GPUmem \n");


    hipEventRecord(start, 0);

    printf("Got past hipEventRecord \n");

    for (int i = 1; i < i_max; i++) {
        printf("Got in for loop \n");
        // execute kernel
        calculate_next<<<ceil((double)t_max/block_size), block_size>>>(
                dev_old + 1, dev_cur + 1, dev_new + 1, t_max - 2, i);

        printf("Calculated for i = %d \n", i);

        hipDeviceSynchronize();

        // switch pointers over
        double *temp = dev_old;
        dev_old = dev_cur;
        dev_cur = dev_new;
        dev_new = temp;
        printf("switched arrays for i = %d \n", i);
    }

    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy results back
    checkCudaCall(hipMemcpy(current_array, dev_cur, t_max * sizeof(double),
            hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(dev_old));
    checkCudaCall(hipFree(dev_cur));
    checkCudaCall(hipFree(dev_new));

    /* You should return a pointer to the array with the final results. */

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("kernel invocation took %f milliseconds\n", elapsedTime);

    return current_array;
}



typedef double (*func_t)(double x);

/*
 * Simple gauss with mu=0, sigma^1=1
 */
double gauss(double x)
{
    return exp((-1 * x * x) / 2);
}


/*
 * Fills a given array with samples of a given function. This is used to fill
 * the initial arrays with some starting data, to run the simulation on.
 *
 * The first sample is placed at array index `offset'. `range' samples are
 * taken, so your array should be able to store at least offset+range doubles.
 * The function `f' is sampled `range' times between `sample_start' and
 * `sample_end'.
 */
void fill(double *array, int offset, int range, double sample_start,
        double sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}


int main(int argc, char *argv[])
{
    double *old, *current, *next;
    int t_max, i_max, block_size;
    timer vectorAddTimer("vector add timer");

    /* Parse commandline args: i_max t_max block_size */
    if (argc < 4) {
        printf("Usage: %s i_max t_max block_size [initial_data]\n", argv[0]);
        printf(" - i_max: number of discrete amplitude points, should be >2\n");
        printf(" - t_max: number of discrete timesteps, should be >=1\n");
        printf(" - block_size: number of threads to use for simulation, "
                "should be >=1\n");
        printf(" - initial_data: select what data should be used for the first "
                "two generation.\n");
        printf("   Available options are:\n");
        printf("    * sin: one period of the sinus function at the start.\n");
        printf("    * sinfull: entire data is filled with the sinus.\n");
        printf("    * gauss: a single gauss-function at the start.\n");
        printf("    * file <2 filenames>: allows you to specify a file with on "
                "each line a float for both generations.\n");

        return EXIT_FAILURE;
    }


    i_max = atoi(argv[1]);
    t_max = atoi(argv[2]);
    block_size = atoi(argv[3]);

    if (i_max < 3) {
        printf("argument error: i_max should be >2.\n");
        return EXIT_FAILURE;
    }
    if (t_max < 1) {
        printf("argument error: t_max should be >=1.\n");
        return EXIT_FAILURE;
    }
    if (block_size < 1) {
        printf("argument error: block_size should be >=1.\n");
        return EXIT_FAILURE;
    }

    printf("Got past argchecking \n");

    /* Allocate and initialize buffers. */
    old = (double *) malloc(i_max * sizeof(double));
    current = (double *) malloc(i_max * sizeof(double));
    next = (double *) malloc(i_max * sizeof(double));

    printf("Got past malloc \n");

    if (old == NULL || current == NULL || next == NULL) {
        fprintf(stderr, "Could not allocate enough memory, aborting.\n");
        return EXIT_FAILURE;
    }

    memset(old, 0, i_max * sizeof(double));
    memset(current, 0, i_max * sizeof(double));
    memset(next, 0, i_max * sizeof(double));

    printf("Got past memset \n");

    /* How should we will our first two generations? */
    if (argc > 4) {
        if (strcmp(argv[4], "sin") == 0) {
            fill(old, 1, i_max/4, 0, 2*3.14, sin);
            fill(current, 2, i_max/4, 0, 2*3.14, sin);
        } else if (strcmp(argv[4], "sinfull") == 0) {
            fill(old, 1, i_max-2, 0, 10*3.14, sin);
            fill(current, 2, i_max-3, 0, 10*3.14, sin);
        } else if (strcmp(argv[4], "gauss") == 0) {
            fill(old, 1, i_max/4, -3, 3, gauss);
            fill(current, 2, i_max/4, -3, 3, gauss);
        } else if (strcmp(argv[4], "file") == 0) {
            if (argc < 7) {
                printf("No files specified!\n");
                return EXIT_FAILURE;
            }
            file_read_double_array(argv[5], old, i_max);
            file_read_double_array(argv[6], current, i_max);
        } else {
            printf("Unknown initial mode: %s.\n", argv[4]);
            return EXIT_FAILURE;
        }
    } else {
        /* Default to sinus. */
        fill(old, 1, i_max/4, 0, 2*3.14, sin);
        fill(current, 2, i_max/4, 0, 2*3.14, sin);
    }

    printf("Got past arrayfilling \n");


    vectorAddTimer.start();

    printf("Got past starting of the timer \n");


    /* Call the actual simulation that should be implemented in simulate.c. */
    simulate(i_max, t_max, block_size, old, current, next);

    printf("Got past simulating \n");

    vectorAddTimer.stop();

    printf("Got past stopping of the timer \n");

    //printf("second timer: %f\n", vectorAddTimer);

    file_write_double_array("result.txt", current, i_max);

    free(old);
    free(current);
    free(next);

    return EXIT_SUCCESS;
}

