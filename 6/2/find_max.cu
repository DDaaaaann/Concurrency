#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "timer.h"
#include <iostream>

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void vectorAddKernel(int n, float* deviceA, int offset) {
    unsigned i = (blockIdx.x * blockDim.x + threadIdx.x) * offset * 2;
    if (i < n && i + offset < n)
        if (deviceA[i] < deviceA[i + offset])
            deviceA[i] = deviceA[i + offset];
}


float vectorMaxCuda(int n, float* a) {
    int threadBlockSize = 512;
    int offset = 1;

    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1.0;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));

    // execute kernels
    hipEventRecord(start, 0);
    for (int i = 1; i < sqrt(n) + 1; i++) {
        vectorAddKernel<<<ceilf(ceilf(n/(float)(2 * offset))/threadBlockSize), threadBlockSize>>>(n, deviceA, offset);
        offset *= 2;
    }
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back

    checkCudaCall(hipFree(deviceA));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;

    return a[0];
}


int main(int argc, char* argv[]) {
    int n = atoi(argv[1]);
    timer vectorAddTimer("vector add timer");
    float* a = new float[n];
    float result = 0.0;
    float check_result = 0.0;


    srand((unsigned)time(0));

    // initialize the vectors.
    for(int i=0; i<n; i++) {
        a[i] = (float)rand()/(float)RAND_MAX;
    }

    vectorAddTimer.start();
    result = vectorMaxCuda(n, a);
    vectorAddTimer.stop();

    cout << vectorAddTimer;

    // verify the resuls

    for(int i=0; i<n; i++) {
        if (a[i] > check_result) {
            check_result = a[i];
        }
    }
    if (check_result != result)
        cout << "error in results! result is " << result << ", but should be " << check_result << endl;

    cout << "results OK!" << endl;

    delete[] a;

    return 0;
}
